
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;
using namespace chrono;

#define GetIndex(mtx, row, col) ((row) * (mtx).width + (col))
#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements = nullptr;
} Matrix;

// Implement matrix multiplication on the CPU
bool MatMulCPU(const Matrix& A, const Matrix& B, Matrix& C) {
    if (A.width != B.height) {
        return false;
    }

    if (C.elements != nullptr) {
        delete[] C.elements;
    }

    C.height = A.height;
    C.width = B.width;
    C.elements = new float[C.width * C.height];

    // Matrix multiplication
    for (int i = 0; i < A.height; ++i) {
        for (int j = 0; j < B.width; ++j) {
            float result = 0;
            for (int e = 0; e < A.width; e++) {
                result += A.elements[GetIndex(A, i, e)] * B.elements[GetIndex(B, e, j)];
            }
            C.elements[GetIndex(C, i, j)] = result;
        }
    }

    return true;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
bool MatMulGPU(const Matrix& A, const Matrix& B, Matrix& C) {
    if (A.width != B.height) {
        return false;
    }

    if (C.elements != NULL) {
        delete[] C.elements;
    }

    C.height = A.height;
    C.width = B.width;
    C.elements = new float[C.width * C.height];

    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((B.width + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                 (A.height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    Matrix d_C;
    /* Your code goes here */
    
    /* End of your code */

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    return true;
}

// Matrix multiplication kernel called by MatMul()
__global__ 
void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    // Each thread computes one element of C
    // by accumulating results into result
    float result = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= C.height || col >= C.width) {
        return;
    }

    /* Your code goes here */

    /* End of your code */
}

int main() {
    const int size = 1 << 11;

    const int N = size; 
    const int M = size;
    const int P = size;

    cout << "Matrix dimensions: " << N << " x " << M << " and " << M << " x " << P << endl;
    cout << "Allocating matrices..." << endl;

    Matrix A, B, result_CPU, result_GPU;
    A.width = N; A.height = M; A.elements = new float[N * M];
    B.width = M; B.height = P; B.elements = new float[M * P];

    for (int i = 0; i < N * M; ++i) {
        A.elements[i] = i % 100 / 1000.0f;
    }
    for (int i = 0; i < M * P; ++i) {
        B.elements[i] = i % 200 / 1000.0f;
    }

    cout << "Start matrix multiplication..." << endl;


    auto start_cpu = high_resolution_clock::now();
    MatMulCPU(A, B, result_CPU);
    auto end_cpu = high_resolution_clock::now();

    duration<float> duration_cpu = end_cpu - start_cpu;
    cout << "Matrix multiplication on CPU completed in " << duration_cpu.count() << " seconds" << endl;


    auto start_gpu = high_resolution_clock::now();
    MatMulGPU(A, B, result_GPU);
    auto end_gpu = high_resolution_clock::now();

    duration<float> duration_gpu = end_gpu - start_gpu;
    cout << "Matrix multiplication on GPU completed in " << duration_gpu.count() << " seconds" << endl;

    // Verify the result
    bool success = true;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < P; ++j) {
            // if (result_CPU.elements[GetIndex(result_CPU, i, j)] != result_GPU.elements[GetIndex(result_CPU, i, j)]) {
            if (abs(result_CPU.elements[GetIndex(result_CPU, i, j)] - result_GPU.elements[GetIndex(result_CPU, i, j)]) > 1e-5) {
                cout << "Results mismatch at index (" << i << ", " << j << "): " 
                     << result_CPU.elements[GetIndex(result_CPU, i, j)] << " != " << result_GPU.elements[GetIndex(result_CPU, i, j)] << endl;
                success = false;
            }
        }
    }

    if (success) {
        cout << "Matrix multiplication completed successfully!" << endl;
    } else {
        cout << "Matrix multiplication completed with errors!" << endl;
    }

    // Free allocated memory
    delete[] A.elements;
    delete[] B.elements;
    delete[] result_CPU.elements;
    delete[] result_GPU.elements;

    return 0;
}